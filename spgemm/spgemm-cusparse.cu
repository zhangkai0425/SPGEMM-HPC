#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "common.h"

const char* version_name = "cuSPARSE SpMV";\

#define CHECK_CUSPARSE(ret) if(ret != HIPSPARSE_STATUS_SUCCESS) { fprintf(stderr, "error %d in line %d\n", ret, __LINE__);}

typedef struct {
    hipsparseHandle_t handle;
    hipsparseSpMatDescr_t matA, matB, matC;
    hipsparseSpGEMMDescr_t gemmDesc;
    void *dBuffer1, *dBuffer2;
    size_t bufferSize1, bufferSize2;

} additional_info_t;

typedef additional_info_t *info_ptr_t;

void preprocess(dist_matrix_t *matA, dist_matrix_t *matB) {
    info_ptr_t p = (info_ptr_t)malloc(sizeof(additional_info_t));
    hipsparseCreate(&p->handle);
    hipsparseCreateCsr(&p->matA, matA->global_m, matA->global_m, matA->global_nnz, matA->gpu_r_pos, matA->gpu_c_idx, matA->gpu_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateCsr(&p->matB, matB->global_m, matB->global_m, matB->global_nnz, matB->gpu_r_pos, matB->gpu_c_idx, matB->gpu_values, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    hipsparseCreateCsr(&p->matC, matA->global_m, matB->global_m, 0, NULL, NULL, NULL, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
    
    hipsparseSpGEMM_createDescr(&p->gemmDesc);


    //hipsparseSetMatIndexBase(p->descrA, HIPSPARSE_INDEX_BASE_ZERO);
    //hipsparseSetMatType(p->descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    matA->additional_info = p;
}

void destroy_additional_info(void *additional_info) {
    info_ptr_t p = (info_ptr_t)additional_info;
    hipsparseSpGEMM_destroyDescr(p->gemmDesc);
    hipsparseDestroySpMat(p->matA);
    hipsparseDestroySpMat(p->matB);
    hipsparseDestroySpMat(p->matC);
    hipsparseDestroy(p->handle);
    hipFree(p->dBuffer1);
    hipFree(p->dBuffer2);
    free(p);
}

void spgemm(dist_matrix_t *mat, dist_matrix_t *matB, dist_matrix_t *matC) {
    int m = mat->global_m, nnz = mat->global_nnz;
    const data_t alpha = 1.0, beta = 0.0;
    info_ptr_t p = (info_ptr_t)mat->additional_info;

    CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(p->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, p->matA, p->matB, &beta, 
        p->matC, HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT, p->gemmDesc, &p->bufferSize1, NULL))
    p->dBuffer1 = NULL;
    hipMalloc( &p->dBuffer1, p->bufferSize1);
    CHECK_CUSPARSE(hipsparseSpGEMM_workEstimation(p->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, p->matA, p->matB, &beta, 
        p->matC, HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT, p->gemmDesc, &p->bufferSize1, p->dBuffer1))

    CHECK_CUSPARSE (hipsparseSpGEMM_compute(p->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, p->matA, p->matB, &beta, 
        p->matC, HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT, p->gemmDesc, &p->bufferSize2, NULL))
    p->dBuffer2 = NULL;
    hipMalloc( &p->dBuffer2, p->bufferSize2);
    hipsparseSpGEMM_compute(p->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, p->matA, p->matB, &beta, 
        p->matC, HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT, p->gemmDesc, &p->bufferSize2, p->dBuffer2);

    int64_t C_row, C_col, C_nnz;
    hipsparseSpMatGetSize(p->matC, &C_row, &C_col, &C_nnz);
    matC->global_m = C_row;
    matC->global_nnz = C_nnz;
    hipMalloc(&matC->gpu_r_pos, (C_row+1)*sizeof(int));
    hipMalloc(&matC->gpu_c_idx, C_nnz*sizeof(int));
    hipMalloc(&matC->gpu_values, C_nnz*sizeof(float));
    matC->r_pos = (int*)malloc(sizeof(int)*(C_row+1));
    matC->c_idx = (int*)malloc(sizeof(int)*C_nnz);
    matC->values = (float*)malloc(sizeof(float)*C_nnz);
    printf("Real nnz = %d \n",C_nnz);

    hipsparseCsrSetPointers(p->matC, matC->gpu_r_pos, matC->gpu_c_idx, matC->gpu_values);
    hipsparseSpGEMM_copy(p->handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, p->matA, p->matB, &beta, 
        p->matC, HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT, p->gemmDesc);
        
    hipMemcpy(matC->r_pos, matC->gpu_r_pos, (C_row+1)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(matC->c_idx, matC->gpu_c_idx, (C_nnz)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(matC->values, matC->gpu_values, (C_nnz)*sizeof(float), hipMemcpyDeviceToHost);

}
